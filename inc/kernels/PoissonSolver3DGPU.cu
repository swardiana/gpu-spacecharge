#include "hip/hip_runtime.h"
#include "PoissonSolver3DGPU.h"
#include <hip/hip_runtime.h>
#include <math.h>

// GPU constant variables
__device__ __constant__ int d_coef_StartPos;
__device__ __constant__ int d_grid_StartPos;
__device__ __constant__ float d_h2;
__device__ __constant__ float d_ih2;
__device__ __constant__ float d_tempRatioZ;


/* GPU kernels start */
__global__ void relaxationGaussSeidelRed
(
	float *VPotential,
	float *RhoChargeDensity,
	const int RRow,
	const int ZColumn,
	const int PhiSlice,
	float *coef1, 
	float *coef2,
	float *coef3, 
	float *coef4
)
{
	int index_x, index_y, index, index_left, index_right, index_up, index_down, index_front, index_back, index_coef;

	index_x = blockIdx.x * blockDim.x + threadIdx.x;
	index_y = blockIdx.y * blockDim.y + threadIdx.y;

	index		= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;
	index_left	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (index_x - 1);
	index_right	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (index_x + 1);
	index_up	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + (index_y - 1) * ZColumn + index_x;
	index_down	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + (index_y + 1) * ZColumn + index_x;
	index_front	= d_grid_StartPos + ((blockIdx.z - 1 + PhiSlice) % PhiSlice) * RRow * ZColumn + index_y * ZColumn + index_x;
	index_back	= d_grid_StartPos + ((blockIdx.z + 1) % PhiSlice) * RRow * ZColumn + index_y * ZColumn + index_x;
	index_coef	= d_coef_StartPos + index_y;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		//calculate red			
		if ((blockIdx.z % 2 == 0 && (index_x + index_y) % 2 == 0) || (blockIdx.z % 2 != 0 && (index_x + index_y) % 2 != 0))
		{			
			VPotential[index] = (coef2[index_coef] * VPotential[index_up] + 
								coef1[index_coef] * VPotential[index_down] + 
								d_tempRatioZ * (VPotential[index_left] + VPotential[index_right]) + 
								coef3[index_coef] * (VPotential[index_front] + VPotential[index_back]) + 
								d_h2 * RhoChargeDensity[index]) * coef4[index_coef];
		}
	}
}

__global__ void relaxationGaussSeidelBlack
(
	float *VPotential,
	float *RhoChargeDensity,
	const int RRow,
	const int ZColumn,
	const int PhiSlice,
	float *coef1, 
	float *coef2,
	float *coef3, 
	float *coef4
)
{
	int index_x, index_y, index, index_left, index_right, index_up, index_down, index_front, index_back, index_coef;

	index_x = blockIdx.x * blockDim.x + threadIdx.x;
	index_y = blockIdx.y * blockDim.y + threadIdx.y;

	index		= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;
	index_left	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (index_x - 1);
	index_right	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (index_x + 1);
	index_up	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + (index_y - 1) * ZColumn + index_x;
	index_down	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + (index_y + 1) * ZColumn + index_x;
	index_front	= d_grid_StartPos + ((blockIdx.z - 1 + PhiSlice) % PhiSlice) * RRow * ZColumn + index_y * ZColumn + index_x;
	index_back	= d_grid_StartPos + ((blockIdx.z + 1) % PhiSlice) * RRow * ZColumn + index_y * ZColumn + index_x;
	index_coef	= d_coef_StartPos + index_y;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		//calculate black		
		if ((blockIdx.z % 2 == 0 && (index_x + index_y) % 2 != 0) || (blockIdx.z % 2 != 0 && (index_x + index_y) % 2 == 0))
		{			
			VPotential[index] = (coef2[index_coef] * VPotential[index_up] + 
								coef1[index_coef] * VPotential[index_down] +								
								d_tempRatioZ * (VPotential[index_left] + VPotential[index_right]) + 
								coef3[index_coef] * (VPotential[index_front] + VPotential[index_back]) + 
								d_h2 * RhoChargeDensity[index]) * coef4[index_coef];
		}
	}
}

__global__ void residueCalculation
(
	float *VPotential,
	float *RhoChargeDensity,
	float *DeltaResidue,
	const int RRow,
	const int ZColumn,
	const int PhiSlice,
	float *coef1, 
	float *coef2,
	float *coef3, 
	float *icoef4
)
{
	int index_x, index_y, index, index_left, index_right, index_up, index_down, index_front, index_back, index_coef;

	index_x = blockIdx.x * blockDim.x + threadIdx.x;
	index_y = blockIdx.y * blockDim.y + threadIdx.y;

	index		= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;
	index_left	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (index_x - 1);
	index_right	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (index_x + 1);
	index_up	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + (index_y - 1) * ZColumn + index_x;
	index_down	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + (index_y + 1) * ZColumn + index_x;
	index_front	= d_grid_StartPos + ((blockIdx.z - 1 + PhiSlice) % PhiSlice)  * RRow * ZColumn + index_y * ZColumn + index_x;
	index_back	= d_grid_StartPos + ((blockIdx.z + 1) % PhiSlice)  * RRow * ZColumn + index_y * ZColumn + index_x;
	index_coef	= d_coef_StartPos + index_y;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		DeltaResidue[index] = d_ih2 * (coef2[index_coef] * VPotential[index_up] +
						coef1[index_coef] * VPotential[index_down] +
						d_tempRatioZ * (VPotential[index_left] + VPotential[index_right]) +
						coef3[index_coef] * (VPotential[index_front] + VPotential[index_back]) -
						icoef4[index_coef] * VPotential[index]) + RhoChargeDensity[index];
	}
}

__global__ void restriction2DHalf
(
	float *RhoChargeDensity,
	float *DeltaResidue,	
	const int RRow,
	const int ZColumn,
	const int PhiSlice
)
{
	int index_x, index_y, index;
	int finer_RRow, finer_ZColumn, finer_grid_StartPos;
	int finer_index_x, finer_index_y, finer_index, finer_index_left, finer_index_right, finer_index_up, finer_index_down;
	
	index_x	= blockIdx.x * blockDim.x + threadIdx.x;
	index_y	= blockIdx.y * blockDim.y + threadIdx.y;
	index	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;

	finer_RRow = 2 * RRow - 1;
	finer_ZColumn = 2 * ZColumn - 1;

	finer_grid_StartPos = d_grid_StartPos - (finer_RRow * finer_ZColumn * PhiSlice);

	finer_index_x = index_x * 2;
	finer_index_y = index_y * 2;

	finer_index			= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + finer_index_y * finer_ZColumn + finer_index_x;
	finer_index_left	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + finer_index_y * finer_ZColumn + (finer_index_x - 1);
	finer_index_right	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + finer_index_y * finer_ZColumn + (finer_index_x + 1);
	finer_index_up		= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y - 1) * finer_ZColumn + finer_index_x;
	finer_index_down	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y + 1) * finer_ZColumn + finer_index_x;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		RhoChargeDensity[index] = 0.5 * DeltaResidue[finer_index] + 
								0.125 * (DeltaResidue[finer_index_left] + DeltaResidue[finer_index_right] + DeltaResidue[finer_index_up] + DeltaResidue[finer_index_down]);
	}
}

__global__ void restriction2DFull
(
	float *RhoChargeDensity,
	float *DeltaResidue,	
	const int RRow,
	const int ZColumn,
	const int PhiSlice
)
{
	int index_x, index_y, index;
	int finer_RRow, finer_ZColumn, finer_grid_StartPos;
	int finer_index_x, finer_index_y, finer_index, finer_index_left, finer_index_right, finer_index_up, finer_index_down;
	int finer_index_up_left, finer_index_up_right, finer_index_down_left, finer_index_down_right;
	
	index_x	= blockIdx.x * blockDim.x + threadIdx.x;
	index_y	= blockIdx.y * blockDim.y + threadIdx.y;
	index	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;

	finer_RRow = 2 * RRow - 1;
	finer_ZColumn = 2 * ZColumn - 1;

	finer_grid_StartPos = d_grid_StartPos - (finer_RRow * finer_ZColumn * PhiSlice);

	finer_index_x = index_x * 2;
	finer_index_y = index_y * 2;

	finer_index			= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + finer_index_y * finer_ZColumn + finer_index_x;
	finer_index_left	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + finer_index_y * finer_ZColumn + (finer_index_x - 1);
	finer_index_right	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + finer_index_y * finer_ZColumn + (finer_index_x + 1);
	finer_index_up		= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y - 1) * finer_ZColumn + finer_index_x;
	finer_index_down	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y + 1) * finer_ZColumn + finer_index_x;
	finer_index_up_left		= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y - 1) * finer_ZColumn + (finer_index_x - 1);
	finer_index_up_right	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y - 1) * finer_ZColumn + (finer_index_x + 1);
	finer_index_down_left	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y + 1) * finer_ZColumn + (finer_index_x - 1);
	finer_index_down_right	= finer_grid_StartPos + blockIdx.z * finer_RRow * finer_ZColumn + (finer_index_y + 1) * finer_ZColumn + (finer_index_x + 1);

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		RhoChargeDensity[index] = 0.25 * DeltaResidue[finer_index] +
								0.125 * (DeltaResidue[finer_index_left] + DeltaResidue[finer_index_right] + DeltaResidue[finer_index_up] + DeltaResidue[finer_index_down]) +
								0.0625 * (DeltaResidue[finer_index_up_left] + DeltaResidue[finer_index_up_right] + DeltaResidue[finer_index_down_left] + DeltaResidue[finer_index_down_right]);
	} else {
		RhoChargeDensity[index] =  DeltaResidue[finer_index];
	}
	

}

__global__ void zeroingVPotential
(
	float *VPotential,
	const int RRow,
	const int ZColumn,
	const int PhiSlice
)
{
	int index_x, index_y, index;

	index_x = blockIdx.x * blockDim.x + threadIdx.x;
	index_y = blockIdx.y * blockDim.y + threadIdx.y;

	index		= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		// zeroing V
		VPotential[index] = 0;
	}

	if (index_x == ZColumn - 2) {
		index_x++;
		index			= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;
		VPotential[index] 	= 0;		
	}
}


__global__ void zeroingBoundaryTopBottom
(
	float *VPotential,
	int RRow,
	int ZColumn,
	int PhiSlice
)
{
	int index_x, index_top, index_bottom;
	
	index_x = blockIdx.x * blockDim.x + threadIdx.x;

	index_top = d_grid_StartPos + blockIdx.z * RRow * ZColumn + 0 * ZColumn + index_x;
	index_bottom = d_grid_StartPos + blockIdx.z * RRow * ZColumn + (ZColumn - 1) * ZColumn + index_x;

	if (index_x < RRow)
	{
		VPotential[index_top] = 0.0;
		VPotential[index_bottom] = 0.0;
	}
}

__global__ void zeroingBoundaryLeftRight
(
	float *VPotential,
	int RRow,
	int ZColumn,
	int PhiSlice
)
{
	int index_y, index_left, index_right;
	
	index_y = blockIdx.x * blockDim.x + threadIdx.x;

	index_left = d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + 0;
	index_right = d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + (RRow - 1);

	if (index_y < ZColumn)
	{
		VPotential[index_left] = 0.0;
		VPotential[index_right] = 0.0;
	}
}

__global__ void prolongation2DHalf
(
	float *VPotential,
	const int RRow,
	const int ZColumn,
	const int PhiSlice
)
{
	int index_x, index_y, index;
	
	int coarser_RRow = (RRow >> 1) + 1;
	int coarser_ZColumn = (ZColumn >> 1) + 1;
	int coarser_grid_StartPos = d_grid_StartPos + RRow * ZColumn * PhiSlice;

	int coarser_index_self;
	int coarser_index_up, coarser_index_down, coarser_index_left, coarser_index_right;	
	int coarser_index_up_left, coarser_index_up_right, coarser_index_down_left, coarser_index_down_right;

	index_x	= blockIdx.x * blockDim.x + threadIdx.x;
	index_y	= blockIdx.y * blockDim.y + threadIdx.y;
	index	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		// x odd, y odd
		if ((index_x % 2 != 0) && (index_y % 2 != 0))
		{
			coarser_index_up_left = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2);
			coarser_index_up_right = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2 + 1);
			coarser_index_down_left = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2 + 1) * coarser_ZColumn + (index_x / 2);
			coarser_index_down_right = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2 + 1) * coarser_ZColumn + (index_x / 2 + 1);

			VPotential[index] += 0.25 * (VPotential[coarser_index_up_left] + VPotential[coarser_index_up_right] + VPotential[coarser_index_down_left] + VPotential[coarser_index_down_right]);
		}
		// x even, y odd
		else if ((index_x % 2 == 0) && (index_y % 2 != 0))
		{
			coarser_index_up = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2);
			coarser_index_down = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2 + 1) * coarser_ZColumn + (index_x / 2);

			VPotential[index] += 0.5 * (VPotential[coarser_index_up] + VPotential[coarser_index_down]);
		}
		// x odd, y even
		else if ((index_x % 2 != 0) && (index_y % 2 == 0))
		{
			coarser_index_left = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2);
			coarser_index_right = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2 + 1);

			VPotential[index] += 0.5 * (VPotential[coarser_index_left] + VPotential[coarser_index_right]);
		}
		// x even, y even
		else
		{
			coarser_index_self = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2)	 * coarser_ZColumn + (index_x / 2);

			VPotential[index] += VPotential[coarser_index_self];
		}
	}
}

__global__ void prolongation2DHalfNoAdd
(
	float *VPotential,
	const int RRow,
	const int ZColumn,
	const int PhiSlice
)
{
	int index_x, index_y, index;
	
	int coarser_RRow = (RRow >> 1) + 1;
	int coarser_ZColumn = (ZColumn >> 1) + 1;
	int coarser_grid_StartPos = d_grid_StartPos + RRow * ZColumn * PhiSlice;

	int coarser_index_self;
	int coarser_index_up, coarser_index_down, coarser_index_left, coarser_index_right;	
	int coarser_index_up_left, coarser_index_up_right, coarser_index_down_left, coarser_index_down_right;

	index_x	= blockIdx.x * blockDim.x + threadIdx.x;
	index_y	= blockIdx.y * blockDim.y + threadIdx.y;
	index	= d_grid_StartPos + blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;

	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		// x odd, y odd
		if ((index_x % 2 != 0) && (index_y % 2 != 0))
		{
			coarser_index_up_left = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2);
			coarser_index_up_right = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2 + 1);
			coarser_index_down_left = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2 + 1) * coarser_ZColumn + (index_x / 2);
			coarser_index_down_right = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2 + 1) * coarser_ZColumn + (index_x / 2 + 1);

			VPotential[index] = 0.25 * (VPotential[coarser_index_up_left] + VPotential[coarser_index_up_right] + VPotential[coarser_index_down_left] + VPotential[coarser_index_down_right]);
		}
		// x even, y odd
		else if ((index_x % 2 == 0) && (index_y % 2 != 0))
		{
			coarser_index_up = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2);
			coarser_index_down = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2 + 1) * coarser_ZColumn + (index_x / 2);

			VPotential[index] = 0.5 * (VPotential[coarser_index_up] + VPotential[coarser_index_down]);
		}
		// x odd, y even
		else if ((index_x % 2 != 0) && (index_y % 2 == 0))
		{
			coarser_index_left = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2);
			coarser_index_right = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2) * coarser_ZColumn + (index_x / 2 + 1);

			VPotential[index] = 0.5 * (VPotential[coarser_index_left] + VPotential[coarser_index_right]);
		}
		// x even, y even
		else
		{
			coarser_index_self = coarser_grid_StartPos + blockIdx.z * coarser_RRow * coarser_ZColumn + (index_y / 2)	 * coarser_ZColumn + (index_x / 2);

			VPotential[index] = VPotential[coarser_index_self];
		}
	}
}


__global__ void errorCalculation
(
	float *VPotentialPrev,
	float *VPotential,
	float *EpsilonError,
	const int RRow,
	const int ZColumn,
	const int PhiSlice
)
{
	int index_x, index_y, index;
	float error;
	float sum_error;

	index_x = blockIdx.x * blockDim.x + threadIdx.x;
	index_y = blockIdx.y * blockDim.y + threadIdx.y;

	index =  blockIdx.z * RRow * ZColumn + index_y * ZColumn + index_x;
	
	if (index_x != 0 && index_x < (ZColumn - 1) && index_y != 0 && index_y < (RRow - 1))
	{
		error = VPotential[index] - VPotentialPrev[index];
		sum_error = error * error;
		__syncthreads();

		atomicAdd( EpsilonError, sum_error );

	}
}
/* GPU kernels end */



/* Error related functions start */
float GetErrorNorm2
(
	float * VPotential,
	float * VPotentialPrev,
	const int rows,
	const int cols,
	float weight
) 
{
	float error = 0.0;	
	float sum_error = 0.0;
	for (int i=0;i<rows;i++)
		for (int j=0;j <cols;j++)
			{
				error = (VPotential[i * cols + j] - VPotentialPrev[i * cols + j]) / weight;
				sum_error  += (error * error);
			}
			
	return sum_error / (rows * cols);
}


float GetAbsMax
(
	float *VPotentialExact,
	int size
)
{
	float mymax = 0.0;
	for (int i=0;i< size;i++) 
		if (abs(VPotentialExact[i]) > mymax) mymax = abs(VPotentialExact[i]); 
	return mymax;
}
/* Error related functions end */

/* Restrict Boundary for FCycle start -- just CPU enough */

void Restrict_Boundary
(
	float *VPotential, 
	const int RRow, 
	const int ZColumn, 
	const int PhiSlice, 
	const int Offset
)
{
	int i,ii,j,jj;		
	int finer_RRow = 2 * RRow - 1;
	int finer_ZColumn = 2 * ZColumn - 1;
	
	int finer_Offset = Offset - (finer_RRow * finer_ZColumn * PhiSlice);
	int sliceStart;
	int finer_SliceStart;

	//printf("(%d,%d,%d) -> (%d,%d,%d)\n",RRow,ZColumn,Offset,finer_RRow,finer_ZColumn,finer_Offset); 
	// do for each slice
	for ( int m = 0;m < PhiSlice;m++)
	{	
		sliceStart = m * (RRow * ZColumn);
		finer_SliceStart = m * (finer_RRow * finer_ZColumn);
		// copy boundary
		for ( j = 0, jj =0; j < ZColumn ; j++,jj+=2) 
		{
			VPotential[Offset + sliceStart + (0 * ZColumn) + j] =
				VPotential[finer_Offset + finer_SliceStart + (0 * finer_ZColumn) + jj];

			VPotential[Offset + sliceStart + ((RRow - 1) * ZColumn) + j] =
				VPotential[finer_Offset + finer_SliceStart + ((finer_RRow -1) * finer_ZColumn) + jj];

		}		
		for ( i = 0, ii =0; i < RRow  ; i++,ii+=2) {
			VPotential[Offset + sliceStart + (i * ZColumn)] =
				VPotential[finer_Offset + finer_SliceStart + (ii * finer_ZColumn)];
			
			VPotential[Offset + sliceStart + (i * ZColumn) + (ZColumn - 1)] =
				VPotential[finer_Offset + finer_SliceStart + (ii * finer_ZColumn) + (finer_ZColumn - 1)];

		}
	}
/**
		// top left (0,0)

		// boundary in top and down
		for ( j = 1, jj =2; j < ZColumn-1 ; j++,jj+=2) 
		{
			VPotential[Offset + sliceStart + (0 * ZColumn) + j] =
				0.5 * VPotential[finer_Offset + finer_SliceStart + (0 * finer_ZColumn) + jj] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + (0 * finer_ZColumn) + jj - 1] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + (0 * finer_ZColumn) + jj + 1];
			
			VPotential[Offset + sliceStart + ((RRow - 1) * ZColumn) + j] =
				0.5 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow -1) * finer_ZColumn) + jj] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow -1) * finer_ZColumn) + jj - 1] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow -1) * finer_ZColumn) + jj + 1];

				 
		}
				
		// boundary in left and right
		for ( i = 1, ii =2; i < RRow - 1 ; i++,ii+=2) {
			VPotential[Offset + sliceStart + (i * ZColumn)] =
				0.5 * VPotential[finer_Offset + finer_SliceStart + (ii * finer_ZColumn)] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + ((ii-1) * finer_ZColumn)] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + ((ii + 1) * finer_ZColumn)];
			
			VPotential[Offset + sliceStart + (i * ZColumn) + (ZColumn - 1)] =
				0.5 * VPotential[finer_Offset + finer_SliceStart + (ii * finer_ZColumn) + jj  + (finer_ZColumn - 1)] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + ((ii -1) * finer_ZColumn) + (finer_ZColumn - 1)] +
				0.25 * VPotential[finer_Offset + finer_SliceStart + ((ii +1) * finer_ZColumn) + (finer_ZColumn - 1)];

		}

		// top left (0,0)

		VPotential[Offset + sliceStart + (0 * ZColumn) + 0] =
			0.5 * VPotential[finer_Offset  + finer_SliceStart] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + (0 * finer_ZColumn) + 1] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + (1 * finer_ZColumn)];
		
		// top right
		VPotential[Offset + sliceStart + (0 * ZColumn) + (ZColumn - 1) ] =
			0.5 * VPotential[finer_Offset + finer_SliceStart  + (0 * finer_ZColumn) + (finer_ZColumn -1) ] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + (0 * finer_ZColumn) + (finer_ZColumn - 2)] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + (1 * finer_ZColumn) + (finer_ZColumn - 1)];

		
		// bottom left
		VPotential[Offset + sliceStart + ((RRow - 1) * ZColumn) + 0] =
			0.5 * VPotential[finer_Offset + finer_SliceStart  + ((finer_RRow - 1) * finer_ZColumn) + 0] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow - 1) * finer_ZColumn) + 1] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow - 2) * finer_ZColumn)  + 0];

		// bottom right
		VPotential[Offset + sliceStart + ((RRow - 1) * ZColumn) + (ZColumn - 1)] =
			0.5 * VPotential[finer_Offset + finer_SliceStart  + ((finer_RRow - 1) * finer_ZColumn) + (finer_ZColumn - 1)] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow - 1) * finer_ZColumn) + (finer_ZColumn - 2)] +
			0.25 * VPotential[finer_Offset + finer_SliceStart + ((finer_RRow - 2) * finer_ZColumn)  + (finer_ZColumn - 1)];	

	}
**/
}

/* Restrict Boundary for FCycle end */

/** Print matrix  **/

void PrintMatrix
(
	float *Mat,
	const int Row, 
	const int Column
)
{
	printf("Matrix (%d,%d)\n",Row,Column);
	for (int i=0;i<Row;i++)
	{
		for (int j=0;j<Column;j++)
		{
			printf("%11.4g ",Mat[i*Column + j]);
		}
		printf("\n");
	}

} 



/* Cycle functions start */
void VCycleSemiCoarseningGPU
(
	float *d_VPotential,
	float *d_RhoChargeDensity,
	float *d_DeltaResidue,
	float *d_coef1,
	float *d_coef2,
	float *d_coef3,
	float *d_coef4,
	float *d_icoef4,
	float gridSizeR,
	float ratioZ,
	float ratioPhi,
	int RRow,
	int ZColumn,
	int PhiSlice,
	int gridFrom,
	int gridTo,
	int nPre,
	int nPost	
)
{
	int grid_RRow;
	int grid_ZColumn;
	int grid_PhiSlice = PhiSlice;
	int grid_StartPos;
	int coef_StartPos;
	int iOne, jOne;
	float h, h2, ih2;
	float tempRatioZ;
	float tempRatioPhi;
	float radius;
	
	// V-Cycle => Finest Grid
	iOne = 1 << (gridFrom - 1); 
	jOne = 1 << (gridFrom - 1);

	//grid_RRow		= ((RRow - 1) / iOne) + 1;
	//grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

	// change accordingly to gridFrom
	grid_StartPos = 0;
	coef_StartPos = 0;


	for (int step = 1; step < gridFrom; step++)
	{
		grid_RRow = ((RRow - 1) / (1 << (step - 1))) + 1;
		grid_ZColumn = ((ZColumn - 1) / (1 << (step - 1))) + 1;
		
		grid_StartPos += grid_RRow * grid_ZColumn * grid_PhiSlice;
		coef_StartPos += grid_RRow;
	}

	grid_RRow		= ((RRow - 1) / iOne) + 1;
	grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;



	// pre-compute constant memory
	h 	= gridSizeR * iOne;
	h2	= h * h;
	ih2	= 1.0 / h2;

	tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
	tempRatioPhi = ratioPhi * iOne * iOne;

	// copy constant to device memory
	hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

	// set kernel grid size and block size
	dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
	dim3 grid_ThreadPerBlock(16, 16);

	// red-black gauss seidel relaxation (nPre times)
	for (int i = 0; i < nPre; i++)
	{
		relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
		relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
	}

	// residue calculation
	residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );

	// V-Cycle => from finer to coarsest grid
	for (int step = gridFrom + 1; step <= gridTo; step++)
	{
		iOne = 1 << (step - 1); 
		jOne = 1 << (step - 1);

		grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;
		coef_StartPos += grid_RRow;

		grid_RRow		= ((RRow - 1) / iOne) + 1;
		grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

		// pre-compute constant memory
		h	= gridSizeR * iOne;
		h2	= h * h;
		ih2	= 1.0 / h2;

		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		// copy constant to device memory
		hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

		// set kernel grid size and block size
		dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
		dim3 grid_ThreadPerBlock(16, 16);

		// restriction
		restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice );

		// zeroing V
		zeroingVPotential<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );

		// zeroing boundaries
		dim3 grid_BlockPerGridTopBottom((grid_RRow < 16) ? 1 : ((grid_RRow / 16) + 1), 1, PhiSlice);
		dim3 grid_BlockPerGridLeftRight((grid_ZColumn < 16) ? 1 : ((grid_ZColumn / 16) + 1), 1, PhiSlice);
		dim3 grid_ThreadPerBlockBoundary(16);

		zeroingBoundaryTopBottom<<< grid_BlockPerGridTopBottom, grid_ThreadPerBlockBoundary >>>( d_VPotential, grid_RRow, grid_ZColumn, PhiSlice );
		zeroingBoundaryLeftRight<<< grid_BlockPerGridLeftRight, grid_ThreadPerBlockBoundary >>>( d_VPotential, grid_RRow, grid_ZColumn, PhiSlice );

		// red-black gauss seidel relaxation (nPre times)
		for (int i = 0; i < nPre; i++)
		{
			relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
			relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
		}

		// residue calculation
		if (step < gridTo)
		{
			residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );

		}
	}

	// V-Cycle => from coarser to finer grid
	for (int step = (gridTo - 1); step >= gridFrom; step--)
	{
		iOne = iOne / 2;
		jOne = jOne / 2;
	
		grid_RRow		= ((RRow - 1) / iOne) + 1;
		grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

		grid_StartPos -= grid_RRow * grid_ZColumn * PhiSlice;
		coef_StartPos -= grid_RRow;
	
		h	= gridSizeR * iOne;
		h2	= h * h;
		ih2	= 1.0 / h2;
	
		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		// copy constant to device memory
		hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

		// set kernel grid size and block size
		dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
		dim3 grid_ThreadPerBlock(16, 16);

		// prolongation
		prolongation2DHalf<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );

		// red-black gauss seidel relaxation (nPost times)
		for (int i = 0; i < nPost; i++)
		{
			relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
			relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
		}
	}
}
/* Cycle functions end */





/*extern function */
extern "C" void PoissonMultigrid3DSemiCoarseningGPUError
(
	float *VPotential, 
	float *RhoChargeDensity,
	const int RRow, 
	const int ZColumn,  
	const int PhiSlice,   
	const int Symmetry,
	float *fparam,
	int *iparam,
	float *errorConv,
	float *errorExact,
	float *VPotentialExact //allocation in the client
)
{
	// variables for CPU memory
	float *temp_VPotential;
	float *VPotentialPrev;
	float *EpsilonError;	

	// variables for GPU memory	
	float *d_VPotential;
	float *d_RhoChargeDensity;
	float *d_DeltaResidue;
	float *d_VPotentialPrev;
	float *d_EpsilonError;
	
	float *d_coef1;
	float *d_coef2;
	float *d_coef3;
	float *d_coef4;
	float *d_icoef4;

	// variables for coefficent calculations
	float *coef1;
	float *coef2;
	float *coef3;
	float *coef4;
	float *icoef4;
	float tempRatioZ;
	float tempRatioPhi;
	float radius;

	int gridFrom;
	int gridTo; 
	int loops;


	// variables passed from ALIROOT
	float gridSizeR		= fparam[0];
	float gridSizePhi	= fparam[1];
	float gridSizeZ		= fparam[2];
	float ratioPhi		= fparam[3];
	float ratioZ		= fparam[4];
	float convErr		= fparam[5];
	float IFCRadius		= fparam[6];
	int nPre		= iparam[0];
	int nPost		= iparam[1];
	int maxLoop		= iparam[2];
	int nCycle		= iparam[3];

	// variables for calculating GPU memory allocation
	int grid_RRow;
	int grid_ZColumn;
	int grid_PhiSlice = PhiSlice;
	int grid_Size = 0;
	int grid_StartPos;
	int coef_Size = 0;
	int coef_StartPos;
	int iOne, jOne;
	float h, h2, ih2;

	// variables for calculating multigrid maximum depth
	int depth_RRow = 0;
	int depth_ZColumn = 0;
	int temp_RRow = RRow;
	int temp_ZColumn = ZColumn;

	// calculate depth for multigrid
	while (temp_RRow >>= 1) depth_RRow++;  
	while (temp_ZColumn >>= 1) depth_ZColumn++;
  
	loops = (depth_RRow > depth_ZColumn) ? depth_ZColumn : depth_RRow;
	loops = (loops > maxLoop) ? maxLoop : loops;

	gridFrom = 1;
	gridTo = loops;

	// calculate GPU memory allocation for multigrid
	for (int step = gridFrom; step <= gridTo; step++)
	{
		grid_RRow = ((RRow - 1) / (1 << (step - 1))) + 1;
		grid_ZColumn = ((ZColumn - 1) / (1 << (step - 1))) + 1;
		
		grid_Size += grid_RRow * grid_ZColumn * grid_PhiSlice;
		coef_Size += grid_RRow;
	}

	// allocate memory for temporary output
	temp_VPotential 		= (float *) malloc(grid_Size * sizeof(float));
	VPotentialPrev = (float *) malloc(RRow * ZColumn * PhiSlice * sizeof(float));
	EpsilonError = (float *) malloc(1 * sizeof(float));


	// allocate memory for relaxation coefficient
	coef1 = (float *) malloc(coef_Size * sizeof(float));
	coef2 = (float *) malloc(coef_Size * sizeof(float));
	coef3 = (float *) malloc(coef_Size * sizeof(float));
	coef4 = (float *) malloc(coef_Size * sizeof(float));
	icoef4 = (float *) malloc(coef_Size * sizeof(float));

	// pre-compute relaxation coefficient
	coef_StartPos = 0;
	iOne = 1 << (gridFrom - 1); 
	jOne = 1 << (gridFrom - 1);
	
	for (int step = gridFrom; step <= gridTo; step++)
	{
		grid_RRow = ((RRow - 1) / iOne) + 1;

		h = gridSizeR * iOne;
		h2 = h * h;
		ih2 = 1.0 / h2;

		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		for (int i = 1; i < grid_RRow - 1; i++)
		{
			radius = IFCRadius + i * h;
			coef1[coef_StartPos + i] = 1.0 + h / (2 * radius);
			coef2[coef_StartPos + i] = 1.0 - h / (2 * radius);
			coef3[coef_StartPos + i] = tempRatioPhi / (radius * radius);
			coef4[coef_StartPos + i] = 0.5 / (1.0 + tempRatioZ + coef3[coef_StartPos + i]);
			icoef4[coef_StartPos + i] = 1.0 / coef4[coef_StartPos + i];
		}
		coef_StartPos += grid_RRow;
		iOne = 2 * iOne;
		jOne = 2 * jOne;
	}

	// device memory allocation
	hipMalloc( &d_VPotential, grid_Size * sizeof(float) );
	hipMalloc( &d_VPotentialPrev, RRow * ZColumn * PhiSlice * sizeof(float) );
	hipMalloc( &d_EpsilonError, 1 * sizeof(float) );	
	hipMalloc( &d_DeltaResidue, grid_Size * sizeof(float) );
	hipMalloc( &d_RhoChargeDensity, grid_Size * sizeof(float) );
	hipMalloc( &d_coef1, coef_Size * sizeof(float) );
	hipMalloc( &d_coef2, coef_Size * sizeof(float) );
	hipMalloc( &d_coef3, coef_Size * sizeof(float) );
	hipMalloc( &d_coef4, coef_Size * sizeof(float) );
	hipMalloc( &d_icoef4, coef_Size * sizeof(float) );

	// set memory to zero
	hipMemset( d_VPotential, 0, grid_Size * sizeof(float) );
	hipMemset( d_DeltaResidue, 0, grid_Size * sizeof(float) );
	hipMemset( d_RhoChargeDensity, 0, grid_Size * sizeof(float) );
	hipMemset( d_VPotentialPrev, 0, RRow * ZColumn * PhiSlice * sizeof(float) );
	hipMemset( d_EpsilonError, 0, 1 * sizeof(float) );


	// copy data from host to device
	hipMemcpy( d_VPotential, VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice ); //check
	hipMemcpy( d_RhoChargeDensity, RhoChargeDensity, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice ); //check
	hipMemcpy( d_coef1, coef1, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef2, coef2, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef3, coef3, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef4, coef4, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_icoef4, icoef4, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_VPotentialPrev, VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice );
	
	// max exact
	
	float maxAbsExact = GetAbsMax(VPotentialExact, RRow * PhiSlice * ZColumn);
	dim3 error_BlockPerGrid((RRow < 16) ? 1 : (RRow / 16), (ZColumn < 16) ? 1 : (ZColumn / 16), PhiSlice);
	dim3 error_ThreadPerBlock(16, 16);		

	
	for (int cycle = 0; cycle < nCycle; cycle++)
	{
		hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
		errorExact[cycle] = GetErrorNorm2(temp_VPotential, VPotentialExact, RRow * PhiSlice,ZColumn, maxAbsExact); 


		VCycleSemiCoarseningGPU(d_VPotential, d_RhoChargeDensity, d_DeltaResidue, d_coef1, d_coef2, d_coef3, d_coef4, d_icoef4, gridSizeR, ratioZ, ratioPhi, RRow, ZColumn, PhiSlice, gridFrom, gridTo, nPre, nPost);
		

		errorCalculation<<< error_BlockPerGrid, error_ThreadPerBlock >>> ( d_VPotentialPrev, d_VPotential, d_EpsilonError, RRow, ZColumn, PhiSlice);

		hipMemcpy( EpsilonError, d_EpsilonError, 1 * sizeof(float), hipMemcpyDeviceToHost );		
		

		errorConv[cycle] = *EpsilonError  / (RRow * ZColumn * PhiSlice);

		if (((*EpsilonError) / (RRow * ZColumn * PhiSlice)) < convErr)
		{
			//errorConv
			nCycle = cycle;
			break;
		}

		hipMemcpy( d_VPotentialPrev, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToDevice );
		hipMemset( d_EpsilonError, 0, 1 * sizeof(float) );

	}
	iparam[3] = nCycle;

//	for (int cycle = 0; cycle < nCycle; cycle++)
//	{
//		hipMemcpy( temp_VPotentialPrev, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );

	
//		VCycleSemiCoarseningGPU(d_VPotential, d_RhoChargeDensity, d_DeltaResidue, d_coef1, d_coef2, d_coef3, d_coef4, d_icoef4, gridSizeR, ratioZ, ratioPhi, RRow, ZColumn, PhiSlice, gridFrom, gridTo, nPre, nPost);
		
//		hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
//		errorConv[cycle] = GetErrorNorm2(temp_VPotential, temp_VPotentialPrev, RRow * PhiSlice, ZColumn, 1.0); 
//		//errorExact[cycle] = GetErrorNorm2(temp_VPotential, VPotentialExact, RRow * PhiSlice,ZColumn, 1.0); 
//	}


	// copy result from device to host
	hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );

	memcpy(VPotential, temp_VPotential, RRow * ZColumn * PhiSlice * sizeof(float));

	// free device memory
	hipFree( d_VPotential );
	hipFree( d_DeltaResidue );
	hipFree( d_RhoChargeDensity );
	hipFree( d_VPotentialPrev );
	hipFree( d_EpsilonError );
	hipFree( d_coef1 );
	hipFree( d_coef2 );
	hipFree( d_coef3 );
	hipFree( d_coef4 );
	hipFree( d_icoef4 );

	// free host memory
	free( coef1 );
	free( coef2 );
	free( coef3 );
	free( coef4 );
	free( icoef4 );
	free( temp_VPotential );
	free( VPotentialPrev );
}



extern "C" void PoissonMultigrid3DSemiCoarseningGPUErrorWCycle
(
	float *VPotential, 
	float *RhoChargeDensity,
	const int RRow, 
	const int ZColumn,  
	const int PhiSlice,   
	const int Symmetry,
	float *fparam,
	int *iparam,
	float *errorConv,
	float *errorExact,
	float *VPotentialExact //allocation in the client
)
{
	// variables for CPU memory
	float *temp_VPotential;
	float *VPotentialPrev;
	float *EpsilonError;		

	// variables for GPU memory	
	float *d_VPotential;
	float *d_RhoChargeDensity;
	float *d_DeltaResidue;
	float *d_coef1;
	float *d_coef2;
	float *d_coef3;
	float *d_coef4;
	float *d_icoef4;
	float *d_VPotentialPrev;
	float *d_EpsilonError;
	

	// variables for coefficent calculations
	float *coef1;
	float *coef2;
	float *coef3;
	float *coef4;
	float *icoef4;
	float tempRatioZ;
	float tempRatioPhi;
	float radius;

	int gridFrom;
	int gridTo; 
	int loops;

	// variables passed from ALIROOT
	float gridSizeR		= fparam[0];
	//float gridSizePhi	= fparam[1];
	//float gridSizeZ		= fparam[2];
	float ratioPhi		= fparam[3];
	float ratioZ		= fparam[4];
	float convErr		= fparam[5];
	float IFCRadius		= fparam[6];
	int nPre	= iparam[0];
	int nPost	= iparam[1];
	int maxLoop	= iparam[2];
	int nCycle	= iparam[3];

	// variables for calculating GPU memory allocation
	int grid_RRow;
	int grid_ZColumn;
	int grid_PhiSlice = PhiSlice;
	int grid_Size = 0;
	int grid_StartPos;
	int coef_Size = 0;
	int coef_StartPos;
	int iOne, jOne;
	float h, h2, ih2;

	// variables for calculating multigrid maximum depth
	int depth_RRow = 0;
	int depth_ZColumn = 0;
	int temp_RRow = RRow;
	int temp_ZColumn = ZColumn;

	// calculate depth for multigrid
	while (temp_RRow >>= 1) depth_RRow++;  
	while (temp_ZColumn >>= 1) depth_ZColumn++;
  
	loops = (depth_RRow > depth_ZColumn) ? depth_ZColumn : depth_RRow;
	loops = (loops > maxLoop) ? maxLoop : loops;

	gridFrom = 1;
	gridTo = loops;

	// calculate GPU memory allocation for multigrid
	for (int step = gridFrom; step <= gridTo; step++)
	{
		grid_RRow = ((RRow - 1) / (1 << (step - 1))) + 1;
		grid_ZColumn = ((ZColumn - 1) / (1 << (step - 1))) + 1;
		
		grid_Size += grid_RRow * grid_ZColumn * grid_PhiSlice;
		coef_Size += grid_RRow;
	}

	// allocate memory for temporary output
	temp_VPotential 		= (float *) malloc(grid_Size * sizeof(float));
	VPotentialPrev = (float *) malloc(RRow * ZColumn * PhiSlice * sizeof(float));
	EpsilonError = (float *) malloc(1 * sizeof(float));


	// allocate memory for relaxation coefficient
	coef1 = (float *) malloc(coef_Size * sizeof(float));
	coef2 = (float *) malloc(coef_Size * sizeof(float));
	coef3 = (float *) malloc(coef_Size * sizeof(float));
	coef4 = (float *) malloc(coef_Size * sizeof(float));
	icoef4 = (float *) malloc(coef_Size * sizeof(float));

	// pre-compute relaxation coefficient
	coef_StartPos = 0;
	iOne = 1 << (gridFrom - 1); 
	jOne = 1 << (gridFrom - 1);
	
	for (int step = gridFrom; step <= gridTo; step++)
	{
		grid_RRow = ((RRow - 1) / iOne) + 1;

		h = gridSizeR * iOne;
		h2 = h * h;
		ih2 = 1.0 / h2;

		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		for (int i = 1; i < grid_RRow - 1; i++)
		{
			radius = IFCRadius + i * h;
			coef1[coef_StartPos + i] = 1.0 + h / (2 * radius);
			coef2[coef_StartPos + i] = 1.0 - h / (2 * radius);
			coef3[coef_StartPos + i] = tempRatioPhi / (radius * radius);
			coef4[coef_StartPos + i] = 0.5 / (1.0 + tempRatioZ + coef3[coef_StartPos + i]);
			icoef4[coef_StartPos + i] = 1.0 / coef4[coef_StartPos + i];
		}
		coef_StartPos += grid_RRow;
		iOne = 2 * iOne;
		jOne = 2 * jOne;
	}

	// device memory allocation
	hipMalloc( &d_VPotential, grid_Size * sizeof(float) );
	hipMalloc( &d_DeltaResidue, grid_Size * sizeof(float) );
	hipMalloc( &d_VPotentialPrev, RRow * ZColumn * PhiSlice * sizeof(float) );
	hipMalloc( &d_EpsilonError, 1 * sizeof(float) );	
		
	hipMalloc( &d_RhoChargeDensity, grid_Size * sizeof(float) );
	hipMalloc( &d_coef1, coef_Size * sizeof(float) );
	hipMalloc( &d_coef2, coef_Size * sizeof(float) );
	hipMalloc( &d_coef3, coef_Size * sizeof(float) );
	hipMalloc( &d_coef4, coef_Size * sizeof(float) );
	hipMalloc( &d_icoef4, coef_Size * sizeof(float) );

	// set memory to zero
	hipMemset( d_VPotential, 0, grid_Size * sizeof(float) );
	hipMemset( d_DeltaResidue, 0, grid_Size * sizeof(float) );
	hipMemset( d_RhoChargeDensity, 0, grid_Size * sizeof(float) );
	hipMemset( d_VPotentialPrev, 0, RRow * ZColumn * PhiSlice * sizeof(float) );
	hipMemset( d_EpsilonError, 0, 1 * sizeof(float) );


	// copy data from host to device
	hipMemcpy( d_VPotential, VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice ); //check
	hipMemcpy( d_RhoChargeDensity, RhoChargeDensity, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice ); //check
	hipMemcpy( d_coef1, coef1, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef2, coef2, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef3, coef3, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef4, coef4, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_icoef4, icoef4, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_VPotentialPrev, VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice );
	
	// max exact	float maxAbsExact = GetAbsMax(VPotentialExact,RRow * PhiSlice * ZColumn);
	float maxAbsExact = GetAbsMax(VPotentialExact, RRow * PhiSlice * ZColumn);
	dim3 error_BlockPerGrid((RRow < 16) ? 1 : (RRow / 16), (ZColumn < 16) ? 1 : (ZColumn / 16), PhiSlice);
	dim3 error_ThreadPerBlock(16, 16);		


	for (int cycle = 0; cycle < nCycle; cycle++)
	{
	/*V-Cycle starts*/

		// error conv		
		//	hipMemcpy( temp_VPotentialPrev, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
		
		hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
		errorExact[cycle] = GetErrorNorm2(temp_VPotential,VPotentialExact,RRow * PhiSlice,ZColumn,maxAbsExact); 


		// V-Cycle => Finest Grid
		iOne = 1 << (gridFrom - 1); 
		jOne = 1 << (gridFrom - 1);

		grid_RRow		= ((RRow - 1) / iOne) + 1;
		grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

		grid_StartPos = 0;
		coef_StartPos = 0;

		// pre-compute constant memory
		h 	= gridSizeR * iOne;
		h2	= h * h;
		ih2	= 1.0 / h2;

		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		// copy constant to device memory
		hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

		// set kernel grid size and block size
		dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
		dim3 grid_ThreadPerBlock(16, 16);

		// red-black gauss seidel relaxation (nPre times)
		for (int i = 0; i < nPre; i++)
		{
			relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
			//hipDeviceSynchronize();
			relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
			//hipDeviceSynchronize();
		}

		// residue calculation
		residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );
		//hipDeviceSynchronize();

		// V-Cycle => from finer to coarsest grid
		for (int step = gridFrom + 1; step <= gridTo; step++)
		{
			iOne = 1 << (step - 1); 
			jOne = 1 << (step - 1);

			grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos += grid_RRow;

			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			// pre-compute constant memory
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;

			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;

			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);

			// restriction
			restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// zeroing V
			zeroingVPotential<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPre times)
			for (int i = 0; i < nPre; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
			}

			// residue calculation
			if (step < gridTo)
			{
				residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );
				//hipDeviceSynchronize();

			}
		}
		/////////// innner w cycle
		/// up one down one

		// up one


		{
			int step = (gridTo - 1);
			iOne = iOne / 2;
			jOne = jOne / 2;
		
			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			grid_StartPos -= grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos -= grid_RRow;
		
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;
		
			tempRatioPhi = ratioPhi * iOne * iOne;
	
			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);
	
		// prolongation
			prolongation2DHalf<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
//			hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPost times)
			for (int i = 0; i < nPost; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
			}
		}

		// down one
		{
			residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );
				
			iOne = iOne * 2; 
			jOne = jOne * 2;

			grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos += grid_RRow;

			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			// pre-compute constant memory
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;

			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;


			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);

			// restriction
			restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// zeroing V
			zeroingVPotential<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPre times)
			for (int i = 0; i < nPre; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
			}
			
		}
		/// end up one down on
		
		/// up two down two
		// up two from gridTo - 1, to gridTo -3
		for (int step = (gridTo - 1); step >= gridTo - 3; step--)
		{
			iOne = iOne / 2;
			jOne = jOne / 2;
		
			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			grid_StartPos -= grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos -= grid_RRow;
		
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;
		
			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;

			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);
	
			// prolongation
			prolongation2DHalf<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
//			hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPost times)
			for (int i = 0; i < nPost; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
			}
		}
		
		// down to from gridTo - 1, to gridTo -3
		for (int step = gridTo - 3; step <= gridTo - 1; step++)
		{
			iOne = iOne * 2; 
			jOne = jOne * 2;

			grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos += grid_RRow;

			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			// pre-compute constant memory
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;

			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;

			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);

			// restriction
			restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// zeroing V
			zeroingVPotential<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPre times)
			for (int i = 0; i < nPre; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
			}

			// residue calculation
			if (step < gridTo)
			{
				residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );
				//hipDeviceSynchronize();

			}
		}

		

		/// up one down one
		{
			int step = (gridTo - 1);
			iOne = iOne / 2;
			jOne = jOne / 2;
		
			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			grid_StartPos -= grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos -= grid_RRow;
		
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;
		
			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;
	
			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);
	
		// prolongation
			prolongation2DHalf<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
//			hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPost times)
			for (int i = 0; i < nPost; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
			}
		}

		// down one
		{
			residueCalculation<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_icoef4 );
				
			iOne = iOne * 2; 
			jOne = jOne * 2;

			grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos += grid_RRow;

			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			// pre-compute constant memory
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;

			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;


			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);

			// restriction
			restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_RhoChargeDensity, d_DeltaResidue, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// zeroing V
			zeroingVPotential<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
			//hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPre times)
			for (int i = 0; i < nPre; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
				//hipDeviceSynchronize();
			}
			
		}
		/// end up one down one

		/////////// end inner w cyle

		// V-Cycle => from coarser to finer grid
		for (int step = (gridTo - 1); step >= gridFrom; step--)
		{
			iOne = iOne / 2;
			jOne = jOne / 2;
		
			grid_RRow		= ((RRow - 1) / iOne) + 1;
			grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

			grid_StartPos -= grid_RRow * grid_ZColumn * PhiSlice;
			coef_StartPos -= grid_RRow;
		
			h	= gridSizeR * iOne;
			h2	= h * h;
			ih2	= 1.0 / h2;
		
			tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
			tempRatioPhi = ratioPhi * iOne * iOne;

			// copy constant to device memory
			hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
			hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

			// set kernel grid size and block size
			dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
			dim3 grid_ThreadPerBlock(16, 16);
	
			// prolongation
			prolongation2DHalf<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );
//			hipDeviceSynchronize();

			// red-black gauss seidel relaxation (nPost times)
			for (int i = 0; i < nPost; i++)
			{
				relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
				relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
//				hipDeviceSynchronize();
			}
		}

	/*V-Cycle ends*/

		errorCalculation<<< error_BlockPerGrid, error_ThreadPerBlock >>> ( d_VPotentialPrev, d_VPotential, d_EpsilonError, RRow, ZColumn, PhiSlice);

		hipMemcpy( EpsilonError, d_EpsilonError, 1 * sizeof(float), hipMemcpyDeviceToHost );		
		

		errorConv[cycle] = *EpsilonError  / (RRow * ZColumn * PhiSlice);

		if (((*EpsilonError) / (RRow * ZColumn * PhiSlice)) < convErr)
		{
			//errorConv
			nCycle = cycle;
			iparam[3] = nCycle;
			break;
		}

		hipMemcpy( d_VPotentialPrev, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToDevice );
		hipMemset( d_EpsilonError, 0, 1 * sizeof(float) );
		
		
		
	}

	hipDeviceSynchronize();
	// copy result from device to host
	hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );

	memcpy(VPotential, temp_VPotential, RRow * ZColumn * PhiSlice * sizeof(float));

	// free device memory
	hipFree( d_VPotential );
	hipFree( d_VPotentialPrev );
	hipFree( d_EpsilonError );


	hipFree( d_DeltaResidue );
	hipFree( d_RhoChargeDensity );
	hipFree( d_coef1 );
	hipFree( d_coef2 );
	hipFree( d_coef3 );
	hipFree( d_coef4 );
	hipFree( d_icoef4 );

	// free host memory
	free( coef1 );
	free( coef2 );
	free( coef3 );
	free( coef4 );
	free( icoef4 );
	free( temp_VPotential );
	//free( temp_VPotentialPrev );
}


/*extern function */
extern "C" void PoissonMultigrid3DSemiCoarseningGPUErrorFCycle
(
	float *VPotential, 
	float *RhoChargeDensity,
	const int RRow, 
	const int ZColumn,  
	const int PhiSlice,   
	const int Symmetry,
	float *fparam,
	int *iparam,
	float *errorConv,
	float *errorExact,
	float *VPotentialExact //allocation in the client
)
{
	// variables for CPU memory
	float *temp_VPotential;
	float *VPotentialPrev;
	float *EpsilonError;		

	// variables for GPU memory	
	float *d_VPotential;
	float *d_RhoChargeDensity;
	float *d_DeltaResidue;
	float *d_coef1;
	float *d_coef2;
	float *d_coef3;
	float *d_coef4;
	float *d_icoef4;
	float *d_VPotentialPrev;
	float *d_EpsilonError;
	

	// variables for coefficent calculations
	float *coef1;
	float *coef2;
	float *coef3;
	float *coef4;
	float *icoef4;
	float tempRatioZ;
	float tempRatioPhi;
	float radius;

	int gridFrom;
	int gridTo; 
	int loops;

	// variables passed from ALIROOT
	float gridSizeR		= fparam[0];
	//float gridSizePhi	= fparam[1];
	//float gridSizeZ		= fparam[2];
	float ratioPhi		= fparam[3];
	float ratioZ		= fparam[4];
	float convErr		= fparam[5];
	float IFCRadius		= fparam[6];
	int nPre	= iparam[0];
	int nPost	= iparam[1];
	int maxLoop	= iparam[2];
	int nCycle	= iparam[3];

	// variables for calculating GPU memory allocation
	int grid_RRow;
	int grid_ZColumn;
	int grid_PhiSlice = PhiSlice;
	int grid_Size = 0;
	int grid_StartPos;
	int coef_Size = 0;
	int coef_StartPos;
	int iOne, jOne;
	float h, h2, ih2;

	// variables for calculating multigrid maximum depth
	int depth_RRow = 0;
	int depth_ZColumn = 0;
	int temp_RRow = RRow;
	int temp_ZColumn = ZColumn;

	// calculate depth for multigrid
	while (temp_RRow >>= 1) depth_RRow++;  
	while (temp_ZColumn >>= 1) depth_ZColumn++;
  
	loops = (depth_RRow > depth_ZColumn) ? depth_ZColumn : depth_RRow;
	loops = (loops > maxLoop) ? maxLoop : loops;

	gridFrom = 1;
	gridTo = loops;

	// calculate GPU memory allocation for multigrid
	for (int step = gridFrom; step <= gridTo; step++)
	{
		grid_RRow = ((RRow - 1) / (1 << (step - 1))) + 1;
		grid_ZColumn = ((ZColumn - 1) / (1 << (step - 1))) + 1;
		
		grid_Size += grid_RRow * grid_ZColumn * grid_PhiSlice;
		coef_Size += grid_RRow;
	}

	// allocate memory for temporary output
	temp_VPotential 		= (float *) malloc(grid_Size * sizeof(float));
	VPotentialPrev = (float *) malloc(grid_Size * sizeof(float));
	EpsilonError = (float *) malloc(1 * sizeof(float));

	

	for (int i=0;i<grid_Size;i++) temp_VPotential[i] = 0.0;


	// allocate memory for relaxation coefficient
	coef1 = (float *) malloc(coef_Size * sizeof(float));
	coef2 = (float *) malloc(coef_Size * sizeof(float));
	coef3 = (float *) malloc(coef_Size * sizeof(float));
	coef4 = (float *) malloc(coef_Size * sizeof(float));
	icoef4 = (float *) malloc(coef_Size * sizeof(float));

	// pre-compute relaxation coefficient
	// restrict boundary
	coef_StartPos = 0;
	grid_StartPos = 0;

	iOne = 1 << (gridFrom - 1); 
	jOne = 1 << (gridFrom - 1);
	
	for (int step = gridFrom; step <= gridTo; step++)
	{
		grid_RRow = ((RRow - 1) / iOne) + 1;
		grid_ZColumn = ((ZColumn - 1) / iOne) + 1;

		h = gridSizeR * iOne;
		h2 = h * h;
		ih2 = 1.0 / h2;

		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		for (int i = 1; i < grid_RRow - 1; i++)
		{
			radius = IFCRadius + i * h;
			coef1[coef_StartPos + i] = 1.0 + h / (2 * radius);
			coef2[coef_StartPos + i] = 1.0 - h / (2 * radius);
			coef3[coef_StartPos + i] = tempRatioPhi / (radius * radius);
			coef4[coef_StartPos + i] = 0.5 / (1.0 + tempRatioZ + coef3[coef_StartPos + i]);
			icoef4[coef_StartPos + i] = 1.0 / coef4[coef_StartPos + i];
		}

		// call restrict boundary
		if (step == gridFrom) {
			// Copy original VPotential to tempPotential
			memcpy(temp_VPotential,     VPotential, RRow * ZColumn * PhiSlice * sizeof(float));
					
		} else 
		{
			Restrict_Boundary(temp_VPotential, grid_RRow, grid_ZColumn, PhiSlice, grid_StartPos);
		}

		
		coef_StartPos += grid_RRow;
		grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;


		iOne = 2 * iOne;
		jOne = 2 * jOne;
	}

	// device memory allocation
	hipMalloc( &d_VPotential, grid_Size * sizeof(float) );
	hipMalloc( &d_DeltaResidue, grid_Size * sizeof(float) );
	hipMalloc( &d_RhoChargeDensity, grid_Size * sizeof(float) );
	hipMalloc( &d_coef1, coef_Size * sizeof(float) );
	hipMalloc( &d_coef2, coef_Size * sizeof(float) );
	hipMalloc( &d_coef3, coef_Size * sizeof(float) );
	hipMalloc( &d_coef4, coef_Size * sizeof(float) );
	hipMalloc( &d_icoef4, coef_Size * sizeof(float) );
	hipMalloc( &d_VPotentialPrev, grid_Size * sizeof(float) );
	hipMalloc( &d_EpsilonError, 1 * sizeof(float) );	
		

	// set memory to zero
	hipMemset( d_VPotential, 0, grid_Size * sizeof(float) );
	hipMemset( d_DeltaResidue, 0, grid_Size * sizeof(float) );
	hipMemset( d_RhoChargeDensity, 0, grid_Size * sizeof(float) );
	hipMemset( d_VPotentialPrev, 0, grid_Size * sizeof(float) );
	hipMemset( d_EpsilonError, 0, 1 * sizeof(float) );

	// set memory to zero
	hipMemset( d_VPotential, 0, grid_Size * sizeof(float) );
	hipMemset( d_DeltaResidue, 0, grid_Size * sizeof(float) );
	hipMemset( d_RhoChargeDensity, 0, grid_Size * sizeof(float) );

	// copy data from host to devicei
	// case of FCycle you need to copy all boundary for all
	hipMemcpy( d_VPotential, temp_VPotential, grid_Size * sizeof(float), hipMemcpyHostToDevice ); //check
//	hipMemcpy( d_VPotential, VPotential, grid_Size * isizeof(float), hipMemcpyHostToDevice ); //check

	hipMemcpy( d_RhoChargeDensity, RhoChargeDensity, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice ); //check
//	hipMemcpy( d_RhoChargeDensity, temp_VPotentialPrev, grid_Size * sizeof(float), hipMemcpyHostToDevice ); //check
	hipMemcpy( d_coef1, coef1, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef2, coef2, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef3, coef3, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_coef4, coef4, coef_Size * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_icoef4, icoef4, coef_Size * sizeof(float), hipMemcpyHostToDevice );
//	hipMemcpy( d_VPotentialPrev, temp_VPotential, grid_Size * sizeof(float), hipMemcpyHostToDevice );

//	hipMemcpy( d_VPotentialPrev, VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyHostToDevice );
	
	// max exact
	
	float maxAbsExact = GetAbsMax(VPotentialExact, RRow * PhiSlice * ZColumn);
	
	

	// init iOne,grid_RRow, grid_ZColumn, grid_StartPos, coef_StartPos
	iOne = 1 << (gridFrom - 1); 
	jOne = 1 << (gridFrom - 1);

	grid_RRow		= ((RRow - 1) / iOne) + 1;
	grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

	grid_StartPos = 0;
	coef_StartPos = 0;


	//// Restrict Boundary and Rho	
	for (int step = gridFrom + 1; step <= gridTo; step++)
	{

		iOne = 1 << (step - 1); 
		jOne = 1 << (step - 1);

		grid_StartPos += grid_RRow * grid_ZColumn * PhiSlice;
		coef_StartPos += grid_RRow;

		grid_RRow		= ((RRow - 1) / iOne) + 1;
		grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

		// pre-compute constant memory
		h	= gridSizeR * iOne;
		h2	= h * h;
		ih2	= 1.0 / h2;

		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		// copy constant to device memory
		hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

		// set kernel grid size and block size
		dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
		dim3 grid_ThreadPerBlock(16, 16);

		// restriction
		restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_RhoChargeDensity, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice );
		
		// restrict boundary (already done in cpu)
///		hipMemcpy( temp_VPotential, d_RhoChargeDensity + grid_StartPos , grid_RRow * grid_ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
//		PrintMatrix(temp_VPotential,grid_RRow * PhiSlice,grid_ZColumn);
		// restriction2DFull<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );

		
	}

	dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
	dim3 grid_ThreadPerBlock(16, 16);


	// relax on the coarsest 
	// red-black gauss seidel relaxation (nPre times)
//	printf("rho\n");
//	hipMemcpy( temp_VPotential, d_RhoChargeDensity + grid_StartPos , grid_RRow * grid_ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
//	PrintMatrix(temp_VPotential,grid_RRow,grid_ZColumn);
	
//	printf("v\n");
//	hipMemcpy( temp_VPotential, d_VPotential + grid_StartPos , grid_RRow * grid_ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
//	PrintMatrix(temp_VPotential,grid_RRow,grid_ZColumn);
	for (int i = 0; i < nPre; i++)
	{
		relaxationGaussSeidelRed<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
		hipDeviceSynchronize();
		relaxationGaussSeidelBlack<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, d_RhoChargeDensity, grid_RRow, grid_ZColumn, grid_PhiSlice, d_coef1, d_coef2, d_coef3, d_coef4 );
		hipDeviceSynchronize();
	}

//	printf("v after relax\n");
//	hipMemcpy( temp_VPotential, d_VPotential + grid_StartPos , grid_RRow * grid_ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
//	PrintMatrix(temp_VPotential,grid_RRow,grid_ZColumn);
	
	// V-Cycle => from coarser to finer grid
	for (int step = gridTo -1 ; step >= gridFrom; step--)
	{
		iOne = iOne / 2;
		jOne = jOne / 2;
	
		grid_RRow		= ((RRow - 1) / iOne) + 1;
		grid_ZColumn	= ((ZColumn - 1) / jOne) + 1;

		grid_StartPos -= grid_RRow * grid_ZColumn * PhiSlice;
		coef_StartPos -= grid_RRow;
	
		h	= gridSizeR * iOne;
		h2	= h * h;
		ih2	= 1.0 / h2;
	
		tempRatioZ = ratioZ * iOne * iOne / (jOne * jOne);
		tempRatioPhi = ratioPhi * iOne * iOne;

		// copy constant to device memory
		hipMemcpyToSymbol(HIP_SYMBOL( d_grid_StartPos), &grid_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_coef_StartPos), &coef_StartPos, 1 * sizeof(int), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_h2), &h2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_ih2), &ih2, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
		hipMemcpyToSymbol(HIP_SYMBOL( d_tempRatioZ), &tempRatioZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );

		

		// set kernel grid size and block size
		dim3 grid_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
		dim3 grid_ThreadPerBlock(16, 16);


		prolongation2DHalfNoAdd<<< grid_BlockPerGrid, grid_ThreadPerBlock >>>( d_VPotential, grid_RRow, grid_ZColumn, grid_PhiSlice );

		

		// just 
		
		// max exact
		hipMemcpy( d_VPotentialPrev + grid_StartPos, d_VPotential + grid_StartPos, grid_RRow * grid_ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToDevice );
				
		float maxAbsExact = GetAbsMax(VPotentialExact, RRow * PhiSlice * ZColumn);
		dim3 error_BlockPerGrid((grid_RRow < 16) ? 1 : (grid_RRow / 16), (grid_ZColumn < 16) ? 1 : (grid_ZColumn / 16), PhiSlice);
		dim3 error_ThreadPerBlock(16, 16);		

		

		for (int cycle = 0; cycle < nCycle; cycle++)
		{

				
			if (step == gridFrom) {
				hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
				errorExact[cycle] = GetErrorNorm2(temp_VPotential, VPotentialExact, RRow * PhiSlice,ZColumn, maxAbsExact); 
			}



			//hipDeviceSynchronize();
			VCycleSemiCoarseningGPU(d_VPotential, d_RhoChargeDensity, d_DeltaResidue, d_coef1, d_coef2, d_coef3, d_coef4, d_icoef4, gridSizeR, ratioZ, ratioPhi, RRow, ZColumn, PhiSlice, step, gridTo, nPre, nPost);
			


				//if (step == gridFrom) {
				//hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );
	
				//errorConv[cycle] = GetErrorNorm2(temp_VPotential, VPotentialPrev, RRow * PhiSlice,ZColumn, 1.0); 

				errorCalculation<<< error_BlockPerGrid, error_ThreadPerBlock >>> ( d_VPotentialPrev + grid_StartPos, d_VPotential + grid_StartPos, d_EpsilonError, grid_RRow, grid_ZColumn, PhiSlice);

				hipMemcpy( EpsilonError, d_EpsilonError, 1 * sizeof(float), hipMemcpyDeviceToHost );		
				
				errorConv[cycle] = *EpsilonError  / (grid_RRow * grid_ZColumn * PhiSlice);

				if (((*EpsilonError) / (RRow * ZColumn * PhiSlice)) < convErr)
				{
					nCycle = cycle;			
					break;
				}

				hipMemcpy( d_VPotentialPrev + grid_StartPos, d_VPotential + grid_StartPos, grid_RRow * grid_ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToDevice );
				hipMemset( d_EpsilonError, 0, 1 * sizeof(float) );
				
		}
		
		
	}

	iparam[3] = nCycle;	

	// copy result from device to host
	hipMemcpy( temp_VPotential, d_VPotential, RRow * ZColumn * PhiSlice * sizeof(float), hipMemcpyDeviceToHost );

	memcpy(VPotential, temp_VPotential, RRow * ZColumn * PhiSlice * sizeof(float));

	// free device memory
	hipFree( d_VPotential );
	hipFree( d_DeltaResidue );
	hipFree( d_RhoChargeDensity );
	hipFree( d_coef1 );
	hipFree( d_coef2 );
	hipFree( d_coef3 );
	hipFree( d_coef4 );
	hipFree( d_icoef4 );

	// free host memory
	free( coef1 );
	free( coef2 );
	free( coef3 );
	free( coef4 );
	free( icoef4 );
	free( temp_VPotential );
	free( VPotentialPrev );
}

