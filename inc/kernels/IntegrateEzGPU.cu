#include "hip/hip_runtime.h"
#include "IntegrateEzGPU.h"

#include <hip/hip_runtime.h>

__device__ __constant__ float d_gridSizeZ;
__device__ __constant__ float d_ezField;
__device__ __constant__ int d_scanSize;

__global__ void integrationCalculation
(
	float *d_arrayofIntEx,
	float *d_arrayofEx	
)
{
	extern __shared__ float temp[];

	int threadIndex = threadIdx.x;	
	int arrayIndex = blockIdx.x * (d_scanSize + 1);
	
	float first, second, last;
	
	int n = blockDim.x * 2;

	int offset = 1;
	
	// load data from input
	float temp_a = d_arrayofEx[arrayIndex + (2 * threadIndex)];
	float temp_b = d_arrayofEx[arrayIndex + (2 * threadIndex + 1)];

	// load last element from array to first variable
	first = d_arrayofEx[arrayIndex + d_scanSize];
	second = d_arrayofEx[arrayIndex + d_scanSize - 1];

/* odd function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 4 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 2 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	
	// save odd-numbered scan to even-numbered array
	d_arrayofIntEx[arrayIndex + (2 * threadIndex + 1)] = ((1.5 * first) + (0.5 * second) + temp[(d_scanSize - 1) - (2 * threadIndex)] - temp_b) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);

/* even function */
	// save data to shared memory flipped
	temp[(d_scanSize - 1) - (2 * threadIndex)] = 2 * temp_a;
	temp[(d_scanSize - 1) - (2 * threadIndex + 1)] = 4 * temp_b;
	
	// scan the array
	for (int d = n >> 1; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadIndex == 0)
	{
		last = temp[n - 1];		
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIndex < d)
		{
			int ai = offset * (2 * threadIndex + 1) - 1;
			int bi = offset * (2 * threadIndex + 2) - 1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadIndex == 0)
	{
		d_arrayofIntEx[arrayIndex + d_scanSize] = 0.0;
		d_arrayofIntEx[arrayIndex] = (first + last - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
	else
	{
		d_arrayofIntEx[arrayIndex + (2 * threadIndex)] = (first + temp[(d_scanSize - 1) - (2 * threadIndex) + 1] - temp_a) * (d_gridSizeZ / 3.0) / (-1 * d_ezField);
	}
}

extern "C" void IntegrateEzGPU 
(
	float *arrayOfIntEx, 
	float *arrayOfEx, 
	const int rows, 
	const int columns,  
	const int phislices, 
	float gridSizeZ, 
	float ezField	
)
{
	// initialize device array
	float *d_arrayofIntEx;
	float *d_arrayofEx;

	// set scan size to columns - 1
	int scanSize = columns - 1;

	std::cout << scanSize << std::endl;

	// set grid size and block size
	dim3 gridSize(rows * phislices);
	dim3 blockSize(scanSize / 2);

	// device memory allocation
	hipMalloc( &d_arrayofIntEx, rows * columns * phislices * sizeof(float) );
	hipMalloc( &d_arrayofEx, rows * columns * phislices * sizeof(float) );

	// copy data from host to device
	hipMemcpy( d_arrayofEx, arrayOfEx, rows * columns * phislices * sizeof(float), hipMemcpyHostToDevice );

	// copy constant to device memory
	hipMemcpyToSymbol(HIP_SYMBOL( d_gridSizeZ), &gridSizeZ, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_ezField), &ezField, 1 * sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( d_scanSize), &scanSize, 1 * sizeof(int), 0, hipMemcpyHostToDevice );

	// run the kernel
	integrationCalculation<<< gridSize, blockSize, 2 * scanSize * sizeof(float) >>>( d_arrayofIntEx, d_arrayofEx );

	// copy result from device to host
	hipMemcpy( arrayOfIntEx, d_arrayofIntEx, rows * columns * phislices * sizeof(float), hipMemcpyDeviceToHost );

	// free device memory
	hipFree( d_arrayofIntEx );
	hipFree( d_arrayofEx );
}

